#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define LINE 100000

void readfile(int num[LINE]){
	int temp;
	int i;
	FILE *fp;
	fp = fopen("number.txt", "r");
	i = 0;
	if(fp == NULL){
		printf("Error loading file!!\n");
		exit(1);
	}else{
		while(!feof(fp)){
			fscanf(fp, "%d", &temp);
			num[i] = temp;
			i++;
		}
	}
	fclose(fp);
}

void printfile(int num[LINE]){
	int i;
	FILE *fp = fopen("update.txt", "w");
	for (i = 0; i < LINE; i++)
		fprintf(fp, "%d   ", num[i]);
	fclose(fp);
}

void copyData(int num[LINE], int num1[LINE]){
	int i;
	for(i = 0; i < LINE; i++)
		num1[i] = num[i];
}

__global__ void even(int *dnum, int n){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	k = k * 2;
	if(k <= n - 2){
		if(dnum[k] > dnum[k + 1]){
			temp = dnum[k];
			dnum[k] = dnum[k + 1];
			dnum[k + 1] = temp;
		}
	}
}

__global__ void odd(int *dnum, int n){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	k = k * 2 + 1;
	if(k <= n - 2){
		if(dnum[k] > dnum[k + 1]){
			temp = dnum[k];
			dnum[k] = dnum[k + 1];
			dnum[k + 1] = temp;
		}
	}
}

void docuda(int *dnum, int line){
	int i;
	for(i = 0; i < line; i++){
		even<<<LINE, 256>>>(dnum, line);
		odd<<<LINE, 256>>>(dnum, line);
	}
}

void cuda(int num[LINE], int num1[LINE]){
	int line, i;
	int *dnum;
	struct timeval tv;
	struct timezone tz;
	double start, end, time, time1, time2, average;
	
	start = 0;
	end = 0;
	time = 0;
	time1 = 0;
	time2 = 0;
	line = 10000;
	average = 0;
	printf("Time execution for parallel bubble sort using CUDA using 100k block and 256 threads\n");
	printf("================================================================================\n");
	printf("   Number of data        1st time       2nd time       3rd time       average   \n");
	printf("================================================================================\n");
	while (line <= LINE){
		for (i = 0; i < 3; i++){
			copyData(num, num1);
			hipMalloc(&dnum, LINE*sizeof(int));
			hipMemcpy(dnum, num, LINE*sizeof(int), hipMemcpyHostToDevice);
			gettimeofday(&tv, &tz);
			start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
			docuda(dnum, line);
			gettimeofday(&tv, &tz);
			end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
			hipMemcpy(num, dnum, LINE*sizeof(int), hipMemcpyDeviceToHost);

			if (i == 0)
				time = end - start;
			else if (i == 1)
				time1 = end - start;
			else if (i == 2)
				time2 = end - start;
		}
		average = (time + time1 + time2) / 3;
		printf("      %i              %fs      %fs      %fs      %fs\n", line, time, time1, time2, average);
		line += 10000;
	}
}

int main(){
	int num[LINE];
	int num1[LINE];
	
	printf("Getting data...\n");
	readfile(num);
	printf("Sorting data...\n\n");
	cuda(num, num1);
	printfile(num);
	printf("\nParallel bubble sort in CUDA sucessfully.\n");
	return 0;
}