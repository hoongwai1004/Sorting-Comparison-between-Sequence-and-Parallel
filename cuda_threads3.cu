#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define LINE 100000

void readfile(int num[LINE]){
	int temp;
	int i;
	FILE *fp;
	fp = fopen("number.txt", "r");
	i = 0;
	if(fp == NULL){
		printf("Error loading file!!\n");
		exit(1);
	}else{
		while(!feof(fp)){
			fscanf(fp, "%d", &temp);
			num[i] = temp;
			i++;
		}
	}
	fclose(fp);
}

void printfile(int num[LINE]){
	int i;
	FILE *fp = fopen("update.txt", "w");
	for (i = 0; i < LINE; i++)
		fprintf(fp, "%d   ", num[i]);
	fclose(fp);
}

void copyData(int num[LINE], int num1[LINE]){
	int i;
	for(i = 0; i < LINE; i++)
		num1[i] = num[i];
}

__global__ void even(int *dnum, int n){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	k = k * 2;
	if(k <= n - 2){
		if(dnum[k] > dnum[k + 1]){
			temp = dnum[k];
			dnum[k] = dnum[k + 1];
			dnum[k + 1] = temp;
		}
	}
}

__global__ void odd(int *dnum, int n){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	k = k * 2 + 1;
	if(k <= n - 2){
		if(dnum[k] > dnum[k + 1]){
			temp = dnum[k];
			dnum[k] = dnum[k + 1];
			dnum[k + 1] = temp;
		}
	}
}

void docuda(int *dnum, int threads, int block){
	int i;
	for(i = 0; i < LINE; i++){
		even<<<block, threads>>>(dnum, LINE);
		odd<<<block, threads>>>(dnum, LINE);
	}
}

void cuda(int num[LINE], int num1[LINE]){
	int threads, block, i;
	int *dnum;
	struct timeval tv;
	struct timezone tz;
	double start, end, time, time1, time2, average;
	
	start = 0;
	end = 0;
	time = 0;
	time1 = 0;
	time2 = 0;
	average = 0;
	threads = 256;
	block = 400;
	printf("Time execution for parallel bubble sort using CUDA using 256 threads based on block size\n");
	printf("====================================================================================================\n");
	printf("   Block size       Number of threads        1st time       2nd time       3rd time       average   \n");
	printf("====================================================================================================\n");
	while (block <= 1400){
		for (i = 0; i < 3; i++){
			copyData(num, num1);
			hipMalloc(&dnum, LINE*sizeof(int));
			hipMemcpy(dnum, num, LINE*sizeof(int), hipMemcpyHostToDevice);
			gettimeofday(&tv, &tz);
			start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
			docuda(dnum, threads, block);
			gettimeofday(&tv, &tz);
			end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
			hipMemcpy(num, dnum, LINE*sizeof(int), hipMemcpyDeviceToHost);

			if (i == 0)
				time = end - start;
			else if (i == 1)
				time1 = end - start;
			else if (i == 2)
				time2 = end - start;
		}
		average = (time + time1 + time2) / 3;
		printf("      %i                 %i                %fs      %fs      %fs      %fs\n", block, threads, time, time1, time2, average);
		block += 100;
	}
}

int main(){
	int num[LINE];
	int num1[LINE];
	
	printf("Getting data...\n");
	readfile(num);
	printf("Sorting data...\n\n");
	cuda(num, num1);
	printfile(num);
	printf("\nParallel bubble sort in CUDA sucessfully.\n");
	return 0;
}