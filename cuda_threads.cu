#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define LINE 100000

void readfile(int num[LINE]){
	int temp;
	int i;
	FILE *fp;
	fp = fopen("number.txt", "r");
	i = 0;
	if(fp == NULL){
		printf("Error loading file!!\n");
		exit(1);
	}else{
		while(!feof(fp)){
			fscanf(fp, "%d", &temp);
			num[i] = temp;
			i++;
		}
	}
	fclose(fp);
}

void printfile(int num[LINE]){
	int i;
	FILE *fp = fopen("update.txt", "w");
	for (i = 0; i < LINE; i++)
		fprintf(fp, "%d   ", num[i]);
	fclose(fp);
}

void copyData(int num[LINE], int num1[LINE]){
	int i;
	for(i = 0; i < LINE; i++)
		num1[i] = num[i];
}

__global__ void even(int *dnum, int n){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	k = k * 2;
	if(k <= n - 2){
		if(dnum[k] > dnum[k + 1]){
			temp = dnum[k];
			dnum[k] = dnum[k + 1];
			dnum[k + 1] = temp;
		}
	}
}

__global__ void odd(int *dnum, int n){
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int temp;
	k = k * 2 + 1;
	if(k <= n - 2){
		if(dnum[k] > dnum[k + 1]){
			temp = dnum[k];
			dnum[k] = dnum[k + 1];
			dnum[k + 1] = temp;
		}
	}
}

void docuda(int *dnum, int threads){
	int i, add;
	add = 0;
	if(LINE % threads != 0)
		add = 1;
	else
		add = 0;
	for(i = 0; i < LINE; i++){
		even<<<LINE / threads + add, threads>>>(dnum, LINE);
		odd<<<LINE / threads + add, threads>>>(dnum, LINE);
	}
}

void cuda(int num[LINE], int num1[LINE]){
	int threads, block, i, add;
	int *dnum;
	struct timeval tv;
	struct timezone tz;
	double start, end, time, time1, time2, average;
	
	start = 0;
	end = 0;
	time = 0;
	time1 = 0;
	time2 = 0;
	average = 0;
	threads = 2;
	block = 0;
	printf("Time execution for parallel bubble sort using CUDA based on threads per block\n");
	printf("====================================================================================================\n");
	printf("   Block size       Number of threads        1st time       2nd time       3rd time       average   \n");
	printf("====================================================================================================\n");
	while (threads <= 1024){
		if(LINE % threads != 0)
			add = 1;
		else
			add = 0;
		block = LINE / threads + add;
		for (i = 0; i < 3; i++){
			copyData(num, num1);
			hipMalloc(&dnum, LINE*sizeof(int));
			hipMemcpy(dnum, num, LINE*sizeof(int), hipMemcpyHostToDevice);
			gettimeofday(&tv, &tz);
			start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
			docuda(dnum, threads);
			gettimeofday(&tv, &tz);
			end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000;
			hipMemcpy(num, dnum, LINE*sizeof(int), hipMemcpyDeviceToHost);

			if (i == 0)
				time = end - start;
			else if (i == 1)
				time1 = end - start;
			else if (i == 2)
				time2 = end - start;
		}
		average = (time + time1 + time2) / 3;
		printf("      %i                 %i                %fs      %fs      %fs      %fs\n", block, threads, time, time1, time2, average);
		threads = threads * 2;
	}
}

int main(){
	int num[LINE];
	int num1[LINE];
	
	printf("Getting data...\n");
	readfile(num);
	printf("Sorting data...\n\n");
	cuda(num, num1);
	printfile(num);
	printf("\nParallel bubble sort in CUDA sucessfully.\n");
	return 0;
}